#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

#include "calling_context_tree.h"

enum CallTraceFlags {
  CALL_TRACE_INST = 1,
  CALL_TRACE_CALL = 2,
  CALL_TRACE_RET = 4
};

typedef struct {
  uint64_t func_addr;
  int g_warp_id;
  int offset;
  // 1: normal, 2: call, 4: ret
  int flags;
} call_trace_t;

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE ((1l << 10) * sizeof(call_trace_t))
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
volatile bool skip_flag = false;

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;

uint64_t bb_index = 0;

/* kernel instruction counter, updated by the GPU threads */
__managed__ uint64_t counter = 0;

/* global control variables for this tool */
uint32_t ker_begin_interval = 0;
uint32_t ker_end_interval = UINT32_MAX;
int verbose = 0;
int count_warp_level = 1;
int exclude_pred_off = 0;

#define CALL_STACK_DEBUG 1

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* instrumentation function that we want to inject, please note the use of
 * 1. "extern "C" __device__ __noinline__" to prevent code elimination by the
 * compiler.
 * 2. NVBIT_EXPORT_FUNC(count_instrs) to notify nvbit the name of the function
 * we want to inject. This name must match exactly the function name */
extern "C" __device__ __noinline__ void count_instrs(uint64_t func_addr,
  int offset,
  int num_instrs,
  int count_warp_level) {
  /* all the active threads will compute the active mask */
  const int active_mask = __ballot(1);
  /* each thread will get a lane id (get_lane_id is in utils/utils.h) */
  const int laneid = get_laneid();
  /* get the id of the first active thread */
  const int first_laneid = __ffs(active_mask) - 1;
  /* count all the active thread */
  const int num_threads = __popc(active_mask);
  /* only the first active thread will perform the atomic */
  if (first_laneid == laneid) {
    if (count_warp_level) {
      atomicAdd((unsigned long long *)&counter, 1 * num_instrs);
    } else {
      atomicAdd((unsigned long long *)&counter, num_threads * num_instrs);
    }

    call_trace_t call_trace;
    call_trace.func_addr = func_addr;
    call_trace.g_warp_id = get_global_warp_id();
    call_trace.offset = offset;
    call_trace.flags = CALL_TRACE_INST;

    channel_dev.push(&call_trace, sizeof(call_trace_t));

    if (CALL_STACK_DEBUG) {
      printf("warp %d at function 0x%lx:0x%x\n", call_trace.g_warp_id,
        call_trace.func_addr, call_trace.offset);
    }
  }
}
NVBIT_EXPORT_FUNC(count_instrs)

extern "C" __device__ __noinline__ void count_pred_off(int predicate,
  int count_warp_level) {
  const int active_mask = __ballot(1);

  const int laneid = get_laneid();

  const int first_laneid = __ffs(active_mask) - 1;

  const int predicate_mask = __ballot(predicate);

  const int mask_off = active_mask ^ predicate_mask;

  const int num_threads_off = __popc(mask_off);
  if (first_laneid == laneid) {
    if (count_warp_level) {
      /* if the predicate mask was off we reduce the count of 1 */
      if (predicate_mask == 0)
        atomicAdd((unsigned long long *)&counter, -1);
    } else {
      atomicAdd((unsigned long long *)&counter, -num_threads_off);
    }
    }
  }
NVBIT_EXPORT_FUNC(count_pred_off)

extern "C" __device__ __noinline__ void trace_call(uint64_t func_addr,
  int offset) {
  const int active_mask = __ballot(1);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  if (first_laneid == laneid) {
    call_trace_t call_trace;
    call_trace.func_addr = func_addr;
    call_trace.g_warp_id = get_global_warp_id();
    call_trace.offset = offset;
    call_trace.flags = CALL_TRACE_CALL;

    channel_dev.push(&call_trace, sizeof(call_trace_t));

    if (CALL_STACK_DEBUG) {
      printf("warp %d call at function 0x%lx:0x%x\n", call_trace.g_warp_id,
        call_trace.func_addr, call_trace.offset);
    }
  }
}
NVBIT_EXPORT_FUNC(trace_call)

extern "C" __device__ __noinline__ void trace_ret(uint64_t func_addr,
  int offset) {
  const int active_mask = __ballot(1);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  if (first_laneid == laneid) {
    call_trace_t call_trace;
    call_trace.func_addr = func_addr;
    call_trace.g_warp_id = get_global_warp_id();
    call_trace.offset = offset;
    call_trace.flags = CALL_TRACE_RET;

    channel_dev.push(&call_trace, sizeof(call_trace_t));

    if (CALL_STACK_DEBUG) {
      printf("warp %d ret at function 0x%lx:0x%x\n", call_trace.g_warp_id,
        call_trace.func_addr, call_trace.offset);
    }
  }
}
NVBIT_EXPORT_FUNC(trace_ret)

  /* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We
   * typically do initializations in this call. In this case for instance we get
   * some environment variables values which we use as input arguments to the tool
   */
  void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    GET_VAR_INT(ker_begin_interval, "KERNEL_BEGIN", 0,
      "Beginning of the kernel launch interval where to apply "
      "instrumentation");
    GET_VAR_INT(
      ker_end_interval, "KERNEL_END", UINT32_MAX,
      "End of the kernel launch interval where to apply instrumentation");
    GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
      "Count warp level or thread level instructions");
    GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
      "Exclude predicated off instruction from count");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
  }

/* nvbit_at_function_first_load() is executed every time a function is loaded
 * for the first time. Inside this call-back we typically get the vector of SASS
 * instructions composing the loaded hipFunction_t. We can iterate on this vector
 * and insert call to instrumentation functions before or after each one of
 * them. */
void nvbit_at_function_first_load(hipCtx_t ctx, hipFunction_t func) {
  uint64_t func_addr = nvbit_get_func_addr(func);
  /* Get the static control flow graph of instruction */
  const CFG_t &cfg = nvbit_get_CFG(ctx, func);
  if (cfg.is_degenerate) {
    printf(
      "Warning: Function %s is degenerated, we can't compute basic "
      "blocks statically",
      nvbit_get_func_name(ctx, func));
  }

  if (verbose) {
    printf("Function %s at 0x%lx\n", nvbit_get_func_name(ctx, func), func_addr);
    /* print */
    int cnt = 0;
    for (auto &bb : cfg.bbs) {
      printf("Basic block id %d - num instructions %ld\n", cnt++,
        bb->instrs.size());
      for (auto &i : bb->instrs) {
        i->print(" ");
      }
    }
  }

  if (verbose) {
    printf("inspecting %s - number basic blocks %ld\n",
      nvbit_get_func_name(ctx, func), cfg.bbs.size());
  }

  /* Iterate on basic block and inject the first instruction */
  for (auto &bb : cfg.bbs) {
    Instr *i = bb->instrs[0];
    /* inject device function */
    nvbit_insert_call(i, "count_instrs", IPOINT_BEFORE);
    nvbit_add_call_arg_const_val64(i, func_addr);
    nvbit_add_call_arg_const_val32(i, i->getOffset());
    nvbit_add_call_arg_const_val32(i, bb->instrs.size());
    /* add count warp level option */
    nvbit_add_call_arg_const_val32(i, count_warp_level);
    if (verbose) {
      i->print("Inject count_instr before - ");
    }

    for (auto *i : bb->instrs) { 
      std::string opcode(i->getOpcode());
      if (opcode.find("CAL") != std::string::npos) {
        /* inject device function */
        nvbit_insert_call(i, "trace_call", IPOINT_BEFORE);
        nvbit_add_call_arg_const_val64(i, func_addr);
        nvbit_add_call_arg_const_val32(i, i->getOffset());
        if (verbose) {
          i->print("Inject count_instr before - ");
        }
      }

      if (opcode.find("RET") != std::string::npos) {
        /* inject device function */
        nvbit_insert_call(i, "trace_ret", IPOINT_BEFORE);
        nvbit_add_call_arg_const_val64(i, func_addr);
        nvbit_add_call_arg_const_val32(i, i->getOffset());
        if (verbose) {
          i->print("Inject count_instr before - ");
        }
      }
    }

    ++bb_index;
  }

  if (exclude_pred_off) {
    /* iterate on instructions */
    for (auto i : nvbit_get_instrs(ctx, func)) {
      /* inject only if instruction has predicate */
      if (i->hasPred()) {
        /* inject function */
        nvbit_insert_call(i, "count_pred_off", IPOINT_BEFORE);
        /* add predicate as argument */
        nvbit_add_call_arg_pred_val(i);
        /* add count warp level option */
        nvbit_add_call_arg_const_val32(i, count_warp_level);
        if (verbose) {
          i->print("Inject count_instr before - ");
        }
      }
    }
  }
}

__global__ void flush_channel() {
  /* push memory access with negative cta id to communicate the kernel is
   * completed */
  call_trace_t call_trace;
  call_trace.func_addr = 0;
  channel_dev.push(&call_trace, sizeof(call_trace_t));

  /* flush channel */
  channel_dev.flush();
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
  const char *name, void *params, hipError_t *pStatus) {
  if (skip_flag) return;

  /* Identify all the possible CUDA launch events */
  if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
    cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
    cbid == API_CUDA_cuLaunchKernel) {
    /* cast params to cuLaunch_params since if we are here we know these are
     * the right parameters type */
    cuLaunch_params *p = (cuLaunch_params *)params;

    if (!is_exit) {
      /* if we are entering in a kernel launch:
       * 1. Lock the mutex to prevent multiple kernels to run concurrently
       * (overriding the counter) in case the user application does that
       * 2. Select if we want to run the instrumented or original
       * version of the kernel
       * 3. Reset the kernel instruction counter */
      pthread_mutex_lock(&mutex);
      if (kernel_id >= ker_begin_interval &&
        kernel_id < ker_end_interval) {
        nvbit_enable_instrumented(ctx, p->f, true);
      } else {
        nvbit_enable_instrumented(ctx, p->f, false);
      }
      counter = 0;
      recv_thread_receiving = true;
    } else {
      /* if we are exiting a kernel launch:
       * 1. Wait until the kernel is completed using
       * hipDeviceSynchronize()
       * 2. Get number of thread blocks in the kernel
       * 3. Print the thread instruction counters
       * 4. Release the lock*/
      CUDA_SAFECALL(hipCtxSynchronize());
      skip_flag = true;
      flush_channel<<<1, 1>>>();
      printf("Launch kernel %d - %s\n",
        kernel_id++, nvbit_get_func_name(ctx, p->f));
      CUDA_SAFECALL(hipCtxSynchronize());
      skip_flag = false;

      tot_app_instrs += counter;
      int num_ctas = 0;
      if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params *p2 = (cuLaunchKernel_params *)params;
        num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
      }
      printf(
        "kernel %d - %s - #thread-blocks %d,  kernel "
        "instructions %ld, total instructions %ld\n",
        kernel_id++, nvbit_get_func_name(ctx, p->f), num_ctas, counter,
        tot_app_instrs);

      /* wait here until the receiving thread has not finished with the
       * current kernel */
      while (recv_thread_receiving) {
        pthread_yield();
      }
      pthread_mutex_unlock(&mutex);
    }
  }
}

void *recv_thread_fun(void *) {
  char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
  CallingContextTree cct;

  while (recv_thread_started) {
    uint32_t num_recv_bytes = 0;
    if (recv_thread_receiving &&
      (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
      0) {
      uint32_t num_processed_bytes = 0;

      if (CALL_STACK_DEBUG) {
        printf("recv %d bytes\n", num_recv_bytes);
      }

      while (num_processed_bytes < num_recv_bytes) {
        call_trace_t *call_trace =
          (call_trace_t *)&recv_buffer[num_processed_bytes];

        /* when we get this cta_id_x it means the kernel has completed
         */
        if (call_trace->func_addr == 0) {
          recv_thread_receiving = false;
          break;
        }

        if (CALL_STACK_DEBUG) {
          std::cout << "recv" << std::endl;
          std::cout << "warp_id: " << call_trace->g_warp_id << " flags: " << call_trace->flags <<
            " func_addr: 0x" << std::hex << call_trace->func_addr << " offset: 0x" << call_trace->offset << std::endl;
        }
        
        if ((call_trace->flags & CALL_TRACE_CALL)) {
          cct.call(call_trace->g_warp_id, call_trace->func_addr, call_trace->offset);
        } else if ((call_trace->flags & CALL_TRACE_RET)) {
          cct.ret(call_trace->g_warp_id);
        } else {
          cct.block(call_trace->g_warp_id, call_trace->func_addr, call_trace->offset);
        }

        num_processed_bytes += sizeof(call_trace_t);
      }
    }
  }

  if (CALL_STACK_DEBUG) {
    std::cout << "Calling context tree: " << std::endl;
    std::cout << cct.to_string() << std::endl;
  }

  free(recv_buffer);
  return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  recv_thread_started = true;
  channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
  pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  if (recv_thread_started) {
    recv_thread_started = false;
    pthread_join(recv_thread, NULL);
  }
}
